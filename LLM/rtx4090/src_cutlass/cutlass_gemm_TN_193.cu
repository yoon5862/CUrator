
#include<iostream>
#include<hip/hip_runtime.h>

#include <unistd.h>
#include<string>
#include<fstream>      

#include "cutlass/cutlass.h"
#include "cutlass/layout/matrix.h"
#include "cutlass/gemm/device/gemm.h"
#include <cutlass/util/host_tensor.h>


float cutlass_strided_bathed_sgemm_0(
    int m, int n, int k,
    float alpha, float const *A, int lda, long long int batch_stride_A,
    float const *B, int ldb, long long int batch_stride_B,
    float *C, int ldc, long long int batch_stride_C,
    float beta, int batch_count, int split_k, int warmup=0
){
    using Gemm = cutlass::gemm::device::Gemm<
                                                    float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::ColumnMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float,
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm86,
                                                    cutlass::gemm::GemmShape<96, 256, 8>,
                                                    cutlass::gemm::GemmShape<48, 128, 8>,
                                                    cutlass::gemm::GemmShape<1, 1, 1>,
                                                    cutlass::epilogue::thread::LinearCombination<float, 1, float, float>,
                                                    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
                                                    2,
                                                    1,
                                                    1,
                                                    true,
                                                    cutlass::arch::OpMultiplyAdd
                                                    >;
    
    Gemm gemm_op;
    
    float total_time;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    
    Gemm::Arguments arguments{
            {m, n, k},
            {A, lda},
            {B, ldb},
            {C, ldc},
            {C, ldc},
            {alpha, beta},
            split_k
        };
        
        int *workspace;
        int workspace_size = ((m + (96 - 1)) / 96) * ((m + (256 - 1)) / 256) * sizeof(int);
        hipMalloc(&workspace, workspace_size);
        hipMemset(workspace, 0, workspace_size);
        
        cutlass::Status status = gemm_op.can_implement(arguments);
        if(status != cutlass::Status::kSuccess) return -1;
        
        status = gemm_op.initialize(arguments, workspace);
        if(status != cutlass::Status::kSuccess) return -1;
    
    hipDeviceSynchronize();
    hipEventRecord(start);
    
    for(int i = 0; i < 20; i++){
        status = gemm_op();
        //workspace.release();
        
        if(status != cutlass::Status::kSuccess){
            hipEventDestroy(start);
            hipEventDestroy(end);
            
            std::fstream dataFile2;
            std::string fileName2 = "./rtx4090/rlt_cutlass_TN/" + std::to_string(batch_count) + "_" +
                                    std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
            if(warmup == 0){
                std::string json2 = "{\"dim\": [[96, 256, 8], [48, 128, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                                    + ", \"sizzle\": 8" + " , \"time\": " + std::to_string(-1) + "}";
                
                dataFile2.open(fileName2, std::ios::app);
                
                dataFile2 << json2 << std::endl;
            }
            hipFree(workspace);
            return -1;
        }
    }
    
    hipEventRecord(end);
    hipEventSynchronize(end);
    
    hipEventElapsedTime(&total_time, start, end);
    
    hipFree(workspace);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipDeviceSynchronize();
    
    std::fstream dataFile;
    
    std::string fileName = "./rtx4090/rlt_cutlass_TN/" + std::to_string(batch_count) + "_" +
                            std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
    if(warmup == 0){
        std::string json = "{\"dim\": [[96, 256, 8], [48, 128, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                            + ", \"sizzle\": 8" + " ,\"time\": " + std::to_string(total_time/20) + "}";
        
        dataFile.open(fileName, std::ios::app);
        
        dataFile << json << std::endl;
    }
    
    return total_time / 20;
    
}
        



int main(int argc, char *argv[]){
    float *rlt = new float[1];

    int M = 64;
    int N = 64;
    int K = 64;
    int Batch = 1;
    int split_k = 1;
    int device_id;
    
    int option;
    while((option = getopt(argc, argv, "m:n:k:b:s:d:")) != -1){
        switch(option){
            case 'm':
                M = std::stoi(optarg);
                break;
            case 'n':
                N = std::stoi(optarg);
                break;
            case 'k':
                K = std::stoi(optarg);
                break;
            case 'b':
                Batch = std::stoi(optarg);
                break;
            case 's':
                split_k = std::stoi(optarg);
                break;
            case 'd':
                device_id = std::stoi(optarg);
                break;    
            case '?':
                break;
        }
    }
    
    hipSetDevice(device_id);
    
    int const lda = K;
    int const ldb = K;
    int const ldc = N;
    
    int const count_A = Batch * M * K;
    int const count_B = Batch * N * K;
    int const count_C = Batch * M * N;
    
    long long int batch_stride_A = static_cast<long long int>(M) * static_cast<long long int>(K);
    long long int batch_stride_B = static_cast<long long int>(K) * static_cast<long long int>(N);
    long long int batch_stride_C = static_cast<long long int>(M) * static_cast<long long int>(N);
    
    float alpha = static_cast<float>(1.0f);
    float beta = static_cast<float>(0.0f);
    
    std::vector<float> host_A(count_A, 1.2f);
    std::vector<float> host_B(count_B, 1.0f);
    std::vector<float> host_C(count_C);
    
    float *A;
    float *B;
    float *C;
    
    hipMalloc(&A, count_A * sizeof(float));
    hipMalloc(&B, count_B * sizeof(float));
    hipMalloc(&C, count_C * sizeof(float));
    
    hipMemcpy(A, host_A.data(), count_A * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B, host_B.data(), count_B * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(C, host_C.data(), count_C * sizeof(float), hipMemcpyHostToDevice);
    
    //warmp up
    for(int i = 0; i < 2; i++){
        cutlass_strided_bathed_sgemm_0(M, N, K, alpha, A, lda, batch_stride_A, B, ldb, batch_stride_B, C, ldc, batch_stride_C, beta, Batch, 1, 1);
    }
    
    rlt[0] = cutlass_strided_bathed_sgemm_0(M, N, K, alpha, A, lda, batch_stride_A, B, ldb, batch_stride_B, C, ldc, batch_stride_C, beta, Batch, split_k);

	
    
    hipFree(A);
    hipFree(B);
    hipFree(C);
    
    return 0;
}
        