#include<iostream>
#include<hip/hip_runtime.h>
#include"./softmax.cuh"
using namespace std;

int main(void){
  const int batch = 2;
  const int M = 512;
  const int N = 512;

  float *input = new float[batch * M * N];
  float *output = new float[batch * M * N];

  //input matrix
  for(int i = 0; i < batch * M * N; i++) input[i] = float(i) * 0.001;
  for(int i = 0; i < batch * M * N; i++) output[i] = 0.0f;


  float *d_in, *d_out;
  hipMalloc(&d_in, batch * M * N * sizeof(float));
  hipMalloc(&d_out, batch * M * N * sizeof(float));

  hipMemcpy(d_in, input, batch * M * N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_out, output, batch * M * N * sizeof(float), hipMemcpyHostToDevice);

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipDeviceSynchronize();

  bool can_implement = cutlass::support::can_implement<2, 16, 4>(batch, M, N);
  if(can_implement == false){
    return 0;
  }
  
  //warmming up gpu
  for(int i = 0; i < 100; i++) cutlass::support::softmaxWarp<float, 1, 2, 16, 4>(d_in, d_out, batch, M, N);

  hipEventRecord(start);
  cutlass::support::softmaxWarp<float, 1, 2, 16, 4>(d_in, d_out, batch, M, N);
  hipEventRecord(end);
  hipEventSynchronize(end);

  float time;
  hipEventElapsedTime(&time, start, end);
  std::cout << time << std::endl;

  return 0;
}